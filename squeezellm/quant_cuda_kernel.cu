#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// half-tensor
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/CUDATensorMethods.cuh>

// atomicAdd for double-precision floating-point numbers on hardware with
// compute capability < 6.0 from:
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(
    double* address,
    double val
) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(
      address_as_ull,
      assumed,
      __double_as_longlong(val + __longlong_as_double(assumed))
    );

  // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

const int BLOCKWIDTH  = 128;
const int BLOCKHEIGHT3 =  12;
const int BLOCKHEIGHT4 =  16;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

__device__ inline int as_int(int i) {
  return *reinterpret_cast<int*>(&i);
}

__global__ void VecQuant3MatMulKernelNUQPerChannel(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    int height,
    int width
);

__global__ void VecQuant4MatMulKernelNUQPerChannel(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    int height,
    int width
);

__global__ void VecQuant3MatMulKernelNUQPerChannelBatched(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    int height,
    int width,
    int batch,
    int vec_height
);

__global__ void VecQuant4MatMulKernelNUQPerChannelBatched(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    int height,
    int width,
    int batch,
    int vec_height
);

void vecquant3matmul_nuq_perchannel_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor lookup_table
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT3 - 1) / BLOCKHEIGHT3,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant3MatMulKernelNUQPerChannel<<<blocks, threads>>>(
    vec.data_ptr<float>(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    lookup_table.data_ptr<float>(),
    height, width
  );
}

// 4-bit matvec kernel (LUT-based)
void vecquant4matmul_nuq_perchannel_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor lookup_table
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant4MatMulKernelNUQPerChannel<<<blocks, threads>>>(
    vec.data_ptr<float>(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    lookup_table.data_ptr<float>(),
    height, width
  );
}

// 3-bit batched matvec kernel (LUT-based)
void vecquant3matmul_nuq_perchannel_batched_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor lookup_table
) {
  int height = mat.size(0);
  int width = mat.size(1);

  int batch = vec.size(0);
  int vec_height = vec.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT3 - 1) / BLOCKHEIGHT3,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant3MatMulKernelNUQPerChannelBatched<<<blocks, threads>>>(
    vec.data_ptr<float>(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    lookup_table.data_ptr<float>(),
    height, width, batch, vec_height
  );
}

// 4-bit batched matvec kernel (LUT-based)
void vecquant4matmul_nuq_perchannel_batched_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor lookup_table
) {
  int height = mat.size(0);
  int width = mat.size(1);

  int batch = vec.size(0);
  int vec_height = vec.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant4MatMulKernelNUQPerChannelBatched<<<blocks, threads>>>(
    vec.data_ptr<float>(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    lookup_table.data_ptr<float>(),
    height, width, batch, vec_height
  );
}

__global__ void VecQuant3MatMulKernelNUQPerChannel(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    int height,
    int width
) {

  int row = BLOCKHEIGHT3 * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ float blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT3) * BLOCKWIDTH + threadIdx.x];

  //Modified dequant block
  __shared__ float deq2[8][BLOCKWIDTH];
  int off = threadIdx.x;
  int column_offset = col * 8;
  for (int val = 0; val < 8; val += 1) {
    int lut_index = column_offset + val;
    deq2[val][off] = lookup_table[lut_index];
  }

  int i = width * row + col;
  int k = 0;

  float res = 0;

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;

  __syncthreads();

  while (k < BLOCKWIDTH) {
    tmp1 = as_unsigned(mat[i]);

    res += deq2[(tmp1 >>  0) & 0x7][off] * blockvec[k + 0];
    res += deq2[(tmp1 >>  3) & 0x7][off] * blockvec[k + 1];
    res += deq2[(tmp1 >>  6) & 0x7][off] * blockvec[k + 2];
    res += deq2[(tmp1 >>  9) & 0x7][off] * blockvec[k + 3];
    res += deq2[(tmp1 >>  12) & 0x7][off] * blockvec[k + 4];
    res += deq2[(tmp1 >>  15) & 0x7][off] * blockvec[k + 5];
    res += deq2[(tmp1 >>  18) & 0x7][off] * blockvec[k + 6];
    res += deq2[(tmp1 >>  21) & 0x7][off] * blockvec[k + 7];
    res += deq2[(tmp1 >>  24) & 0x7][off] * blockvec[k + 8];
    res += deq2[(tmp1 >>  27) & 0x7][off] * blockvec[k + 9];

    i += width;
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
    tmp2 >>= 1;
    res += deq2[(tmp >>  0) & 0x7][off] * blockvec[k + 10];
    k += 11;
    res += deq2[(tmp2 >>  0) & 0x7][off] * blockvec[k + 0];
    res += deq2[(tmp2 >>  3) & 0x7][off] * blockvec[k + 1];
    res += deq2[(tmp2 >>  6) & 0x7][off] * blockvec[k + 2];
    res += deq2[(tmp2 >>  9) & 0x7][off] * blockvec[k + 3];
    res += deq2[(tmp2 >>  12) & 0x7][off] * blockvec[k + 4];
    res += deq2[(tmp2 >>  15) & 0x7][off] * blockvec[k + 5];
    res += deq2[(tmp2 >>  18) & 0x7][off] * blockvec[k + 6];
    res += deq2[(tmp2 >>  21) & 0x7][off] * blockvec[k + 7];
    res += deq2[(tmp2 >>  24) & 0x7][off] * blockvec[k + 8];
    res += deq2[(tmp2 >>  27) & 0x7][off] * blockvec[k + 9];

    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
    tmp1 >>= 2;
    res += deq2[(tmp >>  0) & 0x7][off] * blockvec[k + 10];
    k += 11;
    res += deq2[(tmp1 >>  0) & 0x7][off] * blockvec[k + 0];
    res += deq2[(tmp1 >>  3) & 0x7][off] * blockvec[k + 1];
    res += deq2[(tmp1 >>  6) & 0x7][off] * blockvec[k + 2];
    res += deq2[(tmp1 >>  9) & 0x7][off] * blockvec[k + 3];
    res += deq2[(tmp1 >>  12) & 0x7][off] * blockvec[k + 4];
    res += deq2[(tmp1 >>  15) & 0x7][off] * blockvec[k + 5];
    res += deq2[(tmp1 >>  18) & 0x7][off] * blockvec[k + 6];
    res += deq2[(tmp1 >>  21) & 0x7][off] * blockvec[k + 7];
    res += deq2[(tmp1 >>  24) & 0x7][off] * blockvec[k + 8];
    res += deq2[(tmp1 >>  27) & 0x7][off] * blockvec[k + 9];
    i += width;
    k += 10;
  }

  atomicAdd(&mul[col], res);
}

//4-bit per-channel
__global__ void VecQuant4MatMulKernelNUQPerChannel(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    int height,
    int width
) {

  int row = BLOCKHEIGHT4 * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ float blockvec[BLOCKWIDTH];
  blockvec[threadIdx.x] = vec[(row / BLOCKHEIGHT4) * BLOCKWIDTH + threadIdx.x];

  //Modified dequant block
  __shared__ float deq2[16][BLOCKWIDTH];
  int off = threadIdx.x;
  int column_offset = col * 16;
  for (int val = 0; val < 16; val += 1) {
    int lut_index = column_offset + val;
    deq2[val][off] = lookup_table[lut_index];
  }

  __syncthreads();

  float res = 0;
  int i = width * row + col;
  int k = 0;

  unsigned int tmp;

  while (k < BLOCKWIDTH) {
    tmp = as_unsigned(mat[i]);

    res += deq2[(tmp >>  0) & 0xf][off] * blockvec[k + 0];
    res += deq2[(tmp >>  4) & 0xf][off] * blockvec[k + 1];
    res += deq2[(tmp >>  8) & 0xf][off] * blockvec[k + 2];
    res += deq2[(tmp >>  12) & 0xf][off] * blockvec[k + 3];
    res += deq2[(tmp >>  16) & 0xf][off] * blockvec[k + 4];
    res += deq2[(tmp >>  20) & 0xf][off] * blockvec[k + 5];
    res += deq2[(tmp >>  24) & 0xf][off] * blockvec[k + 6];
    res += deq2[(tmp >>  28) & 0xf][off] * blockvec[k + 7];

    i += width;
    k += 8;
  }

  atomicAdd(&mul[col], res);
}


//batched version (3-bit)
__global__ void VecQuant3MatMulKernelNUQPerChannelBatched(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    int height,
    int width,
    int batch,
    int vec_height
) {

  int row = BLOCKHEIGHT3 * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ float blockvec[BLOCKWIDTH];

  __shared__ float deq2[8][BLOCKWIDTH];
  int off = threadIdx.x;
  int column_offset = col * 8;
  for (int val = 0; val < 8; val += 1) {
    int lut_index = column_offset + val;
    deq2[val][off] = lookup_table[lut_index];
  }

  int i;
  float res;
  int k;

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;

  for (int b = 0; b < batch; ++b){
    //initialize vars
    i = width * row + col;
    res = 0;
    k = 0;

    __syncthreads();
    blockvec[threadIdx.x] = vec[b * vec_height + (row / BLOCKHEIGHT3) * BLOCKWIDTH + threadIdx.x];
    __syncthreads();

    while (k < BLOCKWIDTH) {
      tmp1 = as_unsigned(mat[i]);

      res += deq2[(tmp1 >>  0) & 0x7][off] * blockvec[k + 0];
      res += deq2[(tmp1 >>  3) & 0x7][off] * blockvec[k + 1];
      res += deq2[(tmp1 >>  6) & 0x7][off] * blockvec[k + 2];
      res += deq2[(tmp1 >>  9) & 0x7][off] * blockvec[k + 3];
      res += deq2[(tmp1 >>  12) & 0x7][off] * blockvec[k + 4];
      res += deq2[(tmp1 >>  15) & 0x7][off] * blockvec[k + 5];
      res += deq2[(tmp1 >>  18) & 0x7][off] * blockvec[k + 6];
      res += deq2[(tmp1 >>  21) & 0x7][off] * blockvec[k + 7];
      res += deq2[(tmp1 >>  24) & 0x7][off] * blockvec[k + 8];
      res += deq2[(tmp1 >>  27) & 0x7][off] * blockvec[k + 9];

      i += width;
      tmp2 = as_unsigned(mat[i]);
      tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
      tmp2 >>= 1;
      res += deq2[(tmp >>  0) & 0x7][off] * blockvec[k + 10];
      k += 11;
      res += deq2[(tmp2 >>  0) & 0x7][off] * blockvec[k + 0];
      res += deq2[(tmp2 >>  3) & 0x7][off] * blockvec[k + 1];
      res += deq2[(tmp2 >>  6) & 0x7][off] * blockvec[k + 2];
      res += deq2[(tmp2 >>  9) & 0x7][off] * blockvec[k + 3];
      res += deq2[(tmp2 >>  12) & 0x7][off] * blockvec[k + 4];
      res += deq2[(tmp2 >>  15) & 0x7][off] * blockvec[k + 5];
      res += deq2[(tmp2 >>  18) & 0x7][off] * blockvec[k + 6];
      res += deq2[(tmp2 >>  21) & 0x7][off] * blockvec[k + 7];
      res += deq2[(tmp2 >>  24) & 0x7][off] * blockvec[k + 8];
      res += deq2[(tmp2 >>  27) & 0x7][off] * blockvec[k + 9];

      i += width;
      tmp1 = as_unsigned(mat[i]);
      tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
      tmp1 >>= 2;
      res += deq2[(tmp >>  0) & 0x7][off] * blockvec[k + 10];
      k += 11;
      res += deq2[(tmp1 >>  0) & 0x7][off] * blockvec[k + 0];
      res += deq2[(tmp1 >>  3) & 0x7][off] * blockvec[k + 1];
      res += deq2[(tmp1 >>  6) & 0x7][off] * blockvec[k + 2];
      res += deq2[(tmp1 >>  9) & 0x7][off] * blockvec[k + 3];
      res += deq2[(tmp1 >>  12) & 0x7][off] * blockvec[k + 4];
      res += deq2[(tmp1 >>  15) & 0x7][off] * blockvec[k + 5];
      res += deq2[(tmp1 >>  18) & 0x7][off] * blockvec[k + 6];
      res += deq2[(tmp1 >>  21) & 0x7][off] * blockvec[k + 7];
      res += deq2[(tmp1 >>  24) & 0x7][off] * blockvec[k + 8];
      res += deq2[(tmp1 >>  27) & 0x7][off] * blockvec[k + 9];
      i += width;
      k += 10;
    }

    atomicAdd(&mul[b * width + col], res);
  }
}

//batched version (4-bit)
__global__ void VecQuant4MatMulKernelNUQPerChannelBatched(
    const  float* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ lookup_table,
    int height,
    int width,
    int batch,
    int vec_height
) {

  int row = BLOCKHEIGHT4 * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;
  __shared__ float blockvec[BLOCKWIDTH];

  //Modified dequant block
  __shared__ float deq2[16][BLOCKWIDTH];
  int off = threadIdx.x;
  int column_offset = col * 16;
  for (int val = 0; val < 16; val += 1) {
    int lut_index = column_offset + (val & 0xf);
    deq2[val][off] = lookup_table[lut_index];
  }

  int i;
  float res;
  int k;
  unsigned int tmp;

  for (int b = 0; b < batch; ++b){
    i = width * row + col;
    res = 0;
    k = 0;

    __syncthreads();
    blockvec[threadIdx.x] = vec[b * vec_height + (row / BLOCKHEIGHT4) * BLOCKWIDTH + threadIdx.x];
    __syncthreads();

    while (k < BLOCKWIDTH) {
      tmp = as_unsigned(mat[i]);

      res += deq2[(tmp >>  0) & 0xf][off] * blockvec[k + 0];
      res += deq2[(tmp >>  4) & 0xf][off] * blockvec[k + 1];
      res += deq2[(tmp >>  8) & 0xf][off] * blockvec[k + 2];
      res += deq2[(tmp >>  12) & 0xf][off] * blockvec[k + 3];
      res += deq2[(tmp >>  16) & 0xf][off] * blockvec[k + 4];
      res += deq2[(tmp >>  20) & 0xf][off] * blockvec[k + 5];
      res += deq2[(tmp >>  24) & 0xf][off] * blockvec[k + 6];
      res += deq2[(tmp >>  28) & 0xf][off] * blockvec[k + 7];

      i += width;
      k += 8;
    }

    atomicAdd(&mul[b * width + col], res);
  }
}
